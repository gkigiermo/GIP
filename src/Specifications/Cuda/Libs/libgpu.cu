#include<iostream>
#include<hip/hip_runtime.h>

#include<hip/hip_runtime_api.h>
#include<hipblas.h>

using namespace std;
/*
extern "C++" void GIP_cusparseDcsrmv(double *d_val,int *d_indxCol,int *d_indxRow,double *d_x,double *d_y,int rows,int cols,int nnz,cusparseMatDescr_t *descr,cusparseHandle_t *handle)
{
        double alpha=1.0;
        double beta=0.0;
       cusparseDcsrmv(*handle, CUSPARSE_OPERATION_NON_TRANSPOSE, rows, cols,nnz, &alpha,*descr, d_val, d_indxRow, d_indxCol, d_x, &beta, d_y);
}
*/

// Para reservar memoria Double e Int
extern "C++" void GIP_cudaMallocDouble(double* &vector,int size)
{        
	hipMalloc((void**)&vector,size*sizeof(double));
}

extern "C++" void GIP_cudaMallocInt(int* &vector,int size)
{        
	hipMalloc((void**)&vector,size*sizeof(int));
}

// Para copiar a CPU->GPU Double e Int
extern "C++" void GIP_cudaMemcpyDToGpu(double* h_vect,double* d_vect,int size )
{
		hipMemcpy(d_vect,h_vect,size*sizeof(double),hipMemcpyHostToDevice);
		
}

extern "C++" void GIP_cudaMemcpyIToGpu(int* h_vect,int* d_vect,int size )
{
		hipMemcpy(d_vect,h_vect,size*sizeof(int),hipMemcpyHostToDevice);
		
}
// Para copiar a GPU->CPU Double e Int
extern "C++" void GIP_cudaMemcpyIToCpu(int* h_vect, int* d_vect,int size )
{
		hipMemcpy(h_vect,d_vect,size*sizeof(int),hipMemcpyDeviceToHost);
}

extern "C++" void GIP_cudaMemcpyDToCpu(double* h_vect, double* d_vect,int size )
{
                hipMemcpy(h_vect,d_vect,size*sizeof(double),hipMemcpyDeviceToHost);
}

// Para liberar memoria
extern "C++" void GIP_cudaFree(void* vector)
{
	hipFree(vector);
}

extern "C++" void GIP_cudaGetLastError(){
     hipError_t error;
     error=hipGetLastError();
     if(error!= hipSuccess)
     {
       cout<<" ERROR INSIDE A CUDA FUNCTION: "<<error<<" "<<hipGetErrorString(error)<<endl;
       exit(0);
     }
}

//vec1=vec2
extern "C++" void GIP_cudaDcopy(int size, double* vec1, double* vec2)
{
	hipMemcpy(vec1,vec2,size*sizeof(double),hipMemcpyDeviceToDevice);
}

/* In house implementations */
// y=Ax
__global__ void cudaDcsrspmv(int num_rows, int *rowIndA, int* colIndA,double* valA,double* x, double* y)
{
    int row =blockDim.x*blockIdx.x +threadIdx.x;
    if( row<num_rows)
    {
        double sum =0;
        int row_start= rowIndA[row];
        int row_end= rowIndA[row+1];
        for(int j=row_start;j<row_end;j++)
            sum+=valA[j]*x[colIndA[j]];
        y[row]=sum;
    }
}

extern "C++" void GIP_cudaDCsrSpMV(int num_rows, int *rowIndA, int* colIndA,double* valA,double* x, double* y,int blocks,int threads)
{
    dim3 dimGrid(blocks,1,1);
    dim3 dimBlock(threads,1,1);
    hipFuncSetCacheConfig( reinterpret_cast<const void*>(cudaDcsrspmv), hipFuncCachePreferL1 ); // para asignar 48KB a cache en el kernel csr
    cudaDcsrspmv<<<dimGrid,dimBlock>>>(num_rows,rowIndA,colIndA,valA,x,y);

}

// y=alpha*Ax+beta*y
__global__ void cudaDcsrspmvab(int num_rows, int *rowIndA, int* colIndA,double* valA,double* x, double* y, double alpha,double beta)
{
    int row =blockDim.x*blockIdx.x +threadIdx.x;
    if( row<num_rows)
    {
        double sum =0;
        int row_start= rowIndA[row];
        int row_end= rowIndA[row+1];
        for(int j=row_start;j<row_end;j++)
            sum+=valA[j]*x[colIndA[j]];
        y[row]=alpha*sum+beta*y[row];
    }
}

extern "C++" void GIP_cudaDCsrSpMV(int num_rows, int *rowIndA, int* colIndA,double* valA,double* x, double* y,double alpha,double beta,int blocks,int threads)
{
    dim3 dimGrid(blocks,1,1);
    dim3 dimBlock(threads,1,1);
    hipFuncSetCacheConfig( reinterpret_cast<const void*>(cudaDcsrspmvab), hipFuncCachePreferL1 ); // para asignar 48KB a cache en el kernel csr
    cudaDcsrspmvab<<<dimGrid,dimBlock>>>(num_rows,rowIndA,colIndA,valA,x,y,alpha,beta);

}


// y=ax+y
__global__ void cudaDaxpy(int n, double* vec1, double *vec2, double alpha)
{
    int row =blockDim.x*blockIdx.x +threadIdx.x;
    if(row<n)
    {
        vec1[row]+=alpha*vec2[row];
    }
}

extern "C++" void GIP_cudaDaxpy(int n, double* vec1, double *vec2,double alpha,int blocks, int threads)
{

    dim3 dimGrid(blocks,1,1);
    dim3 dimBlock(threads,1,1);
    cudaDaxpy<<<dimGrid,dimBlock>>>(n,vec1,vec2,alpha);
}


//y=ax+by
__global__ void cudaDaxpby(int n, double* vec1, double *vec2, double alpha,double beta)
{
    int row =blockDim.x*blockIdx.x +threadIdx.x;
    if(row<n)
    {
        vec1[row]=alpha*vec2[row]+beta*vec1[row];
    }
}

extern "C++" void GIP_cudaDaxpy(int n, double* vec1, double *vec2,double alpha,double beta,int blocks, int threads)
{

    dim3 dimGrid(blocks,1,1);
    dim3 dimBlock(threads,1,1);
   cudaDaxpby<<<dimGrid,dimBlock>>>(n,vec1,vec2,alpha,beta);
}

__global__ void cudaDdot(double *g_idata1, double *g_idata2, double *g_odata, unsigned int n)
{
    extern __shared__ double sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

    double mySum = (i < n) ? g_idata1[i]*g_idata2[i] : 0;


    if (i + blockDim.x < n)
        mySum += g_idata1[i+blockDim.x]*g_idata2[i+blockDim.x];
    
    sdata[tid] = mySum;
    __syncthreads();

    for (unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
        if (tid < s)
            sdata[tid] = mySum = mySum + sdata[tid + s];

        __syncthreads();
    }

    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


extern "C++" double GIP_cudaDdot(int n, double* vec1,double* vec2,double* h_temp,double* d_temp, int blocks,int threads)
{

    dim3 dimGrid(blocks,1,1);
    dim3 dimBlock(threads,1,1);
    cudaDdot<<<dimGrid,dimBlock,threads*sizeof(double)>>>(vec1,vec2,d_temp,n);


    hipMemcpy(h_temp, d_temp, blocks * sizeof(double), hipMemcpyDeviceToHost);

    double sum=0;
    for(int i=0;i<blocks;i++)
    {
        sum+=h_temp[i];
    }
    return sum;
}


extern "C++" void GIP_cudaEventCreate(hipEvent_t& evet)
{
	hipEventCreate(&evet);
}

extern "C++" void GIP_cudaEventRecord(hipEvent_t& evet)
{
	hipEventRecord(evet,0);
}

extern "C++" void GIP_cudaEventRecord(hipEvent_t& evet, hipStream_t& streamt)
{
	hipEventRecord(evet,streamt);
}

extern "C++" void GIP_cudaEventSynchronize(hipEvent_t& evet)
{
	hipEventSynchronize(evet);
}

extern "C++" float GIP_cudaEventElapsedTime(hipEvent_t& start,hipEvent_t &stop)
{
	float tpo;
	hipEventElapsedTime(&tpo,start,stop);
	return tpo;
}

extern "C++" void GIP_cudaEventDestroy(hipEvent_t& evet)
{
	hipEventDestroy(evet);
}

extern "C++" void GIP_cudaSetDevice(int numDev)
{
	hipSetDevice(numDev);
}


__global__ void getCFL(double *u, double* v,double *w,double *dxs,double *g_odata, unsigned int n,double gamma, double rho)
{
    // double *sdata = SharedMemory<double>();
    extern __shared__ double sdata[];
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

    double dt=1e10;

    double convCoef = 0.25;

    double up=(i<n) ? fabs(u[i]) :0;
    double vp=(i<n) ? fabs(v[i]) :0;
    double wp=(i<n) ? fabs(w[i]) :0;
    double dx=(i<n) ? dxs[i] :0;

    double normiVel= (up>vp) ? up : vp;
    normiVel= (wp>normiVel) ? wp : normiVel;
    if(!(dx/normiVel < 1e-17))
    {
        if(dt>convCoef*dx/(normiVel))
            dt= convCoef*dx/(normiVel);
    }
    if(!(dx*dx*rho/gamma  < 1e-17))
    {
        if(dt>0.2*dx*dx*rho/gamma)
            dt=0.2*dx*dx*rho/gamma;
    }

    if (i + blockDim.x < n)
    {

        up=fabs(u[i+blockDim.x]) ;
        vp=fabs(v[i+blockDim.x]) ;
        wp=fabs(w[i+blockDim.x]) ;
        dx=dxs[i+blockDim.x] ;

        normiVel= (up>vp) ? up : vp;
        normiVel= (wp>normiVel) ? wp : normiVel;
        if(!(dx/normiVel < 1e-17))
        {
            if(dt>convCoef*dx/(normiVel))
                dt= convCoef*dx/(normiVel);
        }
        if(!(dx*dx*rho/gamma  < 1e-17))
        {
            if(dt>0.2*dx*dx*rho/gamma)
                dt=0.2*dx*dx*rho/gamma;
        }

    } 
    sdata[tid] = dt;
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
        if (tid < s)
        {
           sdata[tid] = dt = (sdata[tid + s]<dt) ? sdata[tid + s] : dt;
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


extern "C++" double GIP_gpuGetCFL(double* d_u,double* d_v,double* d_w,double* d_dxs,double* h_odata,double* d_odata,int n,double gamma,double rho, int threads,int blocks)
{

            dim3 dimGrid(blocks,1,1);
            dim3 dimBlock(threads,1,1);// (1024,1024,64)
            getCFL<<<dimGrid,dimBlock,threads*sizeof(double)>>>(d_u,d_v,d_w,d_dxs,d_odata,n,gamma,rho);

           
            hipMemcpy(h_odata, d_odata, blocks * sizeof(double), hipMemcpyDeviceToHost);    

            double min=1e10;
            for(int i=0;i<blocks;i++)
            {
                     min=(min>h_odata[i]) ? h_odata[i] : min;
            }

            
            min=(0.8*min>1e-1)? 1e-1 :0.8*min;
            return min;
}


__global__ void normi(double *g_idata, double *g_odata, unsigned int n)
{
   // double *sdata = SharedMemory<double>();
    extern __shared__ double sdata[];
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

    double myMax = (i < n) ? fabs(g_idata[i]) : 0;

    if (i + blockDim.x < n)
    {
        double next =fabs( g_idata[i+blockDim.x]);
        myMax=(next>myMax) ? next : myMax;
    }
    // mySum += g_idata[i+blockDim.x];

    sdata[tid] = myMax;
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
        if (tid < s)
        {

           sdata[tid] = myMax = (sdata[tid + s]>myMax) ? sdata[tid + s] : myMax;

           // sdata[tid] = mySum = mySum + sdata[tid + s];
        }

        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


 extern "C++" double GIP_gpuNormi(double* d_val,double* h_odata,double* d_odata,int n, int threads,int blocks)
{

            dim3 dimGrid(blocks,1,1);
            dim3 dimBlock(threads,1,1);// (1024,1024,64)
            normi<<<dimGrid,dimBlock,threads*sizeof(double)>>>(d_val,d_odata,n);

           
            hipMemcpy(h_odata, d_odata, blocks * sizeof(double), hipMemcpyDeviceToHost);    

            double max=0;
            for(int i=0;i<blocks;i++)
            {
                     max=(max<h_odata[i]) ? h_odata[i] : max;
            }
            return max;
}
